#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>

//testing commit

//ensure that your code is safeguarded against segmentstion faults etc...
__global__ void cypher_thread(char * t_input, char * t_output, int length){
    int idx = threadIdx.x;

    if(idx < length){
        char c = t_input[idx];
        t_output[idx] = c-1;
    }
}

int main(){
    //initialize test message
    const char program_input[] = "Ifmmp-!J!bn!b!tuvefou!ifsf!jo!uif!Dpnqvufs!Tdjfodf!Efqu/!J!kvtu!xboufe!up!dpohsbuvmbuf!zpv!po!zpvs!ofx!qptjujpo!bt!Dibjs!pg!uif!Efqbsunfou/!!Cftu!xjtift/";
    int length = strlen(program_input);
    int size = length * sizeof(char);
    char program_output[length];

    //declare GPU memory pointers
    char * t_input;
    char * t_output;

    //allocate memory on GPU
    hipMalloc((void **)&t_input, size);
    hipMalloc((void **)&t_output, size);

    //transfer info to GPU
    hipMemcpy(t_input, program_input, size, hipMemcpyHostToDevice);

    //kernel
    cypher_thread<<<1, length>>>(t_input, t_output, length);

    //get result from GPU
    hipMemcpy(program_output, t_output, size, hipMemcpyDeviceToHost);

    //print output
    for(int i = 0; i < length; i++){
        printf("%c", program_output[i]);
    }

    //free gpu memory
    hipFree(t_input);
    hipFree(t_output);

    return 0;
}
